#include "hip/hip_runtime.h"
#include"linear.h"
#include <hip/hip_fp16.h>


cublasWrapper::cublasWrapper(hipblasHandle_t handle)
{
    hipblasCreate(&handle_);
}

void cublasWrapper::sgemm(hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    hipblasSgemm(handle_, transa, transb, m, n, k, &alpha, A, lda, B, ldb, &beta, C, ldc);
}

void cublasWrapper::hgemm(hipblasOperation_t transa, hipblasOperation_t transb, int m, int n, int k, 
                              const __half *A, int lda, const __half *B, int ldb, 
                               __half *C, int ldc, float alpha, float beta) {
    hipblasGemmEx(handle_, transa, transb, m, n, k, 
                 &alpha, A, HIP_R_16F, lda, 
                 B, HIP_R_16F, ldb, 
                 &beta, C, HIP_R_16F, ldc, 
                 HIP_R_16F, HIPBLAS_GEMM_DEFAULT);
}

void cublasWrapper::sgemm_strided_batched(
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const float* A, int lda, long long int strideA,
    const float* B, int ldb, long long int strideB,
    float* C, int ldc, long long int strideC,
    float alpha, float beta, int batchCount)
{
    hipblasGemmStridedBatchedEx(
        handle_, transa, transb,
        m, n, k,
        &alpha,
        A, HIP_R_32F, lda, strideA,
        B, HIP_R_32F, ldb, strideB,
        &beta,
        C, HIP_R_32F, ldc, strideC,
        batchCount,
        HIP_R_32F,
        HIPBLAS_GEMM_DEFAULT);
}

void cublasWrapper::hgemm_strided_batched(
    hipblasOperation_t transa, hipblasOperation_t transb,
    int m, int n, int k,
    const __half* A, int lda, long long int strideA,
    const __half* B, int ldb, long long int strideB,
    __half* C, int ldc, long long int strideC,
    float alpha, float beta, int batchCount)
{
    hipblasGemmStridedBatchedEx(
        handle_, transa, transb,
        m, n, k,
        &alpha,
        A, HIP_R_16F, lda, strideA,
        B, HIP_R_16F, ldb, strideB,
        &beta,
        C, HIP_R_16F, ldc, strideC,
        batchCount,
        HIP_R_16F,
        HIPBLAS_GEMM_DEFAULT);
}

cublasWrapper::~cublasWrapper()
{
    hipblasDestroy(handle_);
}



